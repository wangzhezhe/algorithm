#include "hip/hip_runtime.h"
#include "definitions.cuh"

//Performs CFD calculation on global memory. This code does not use any advance optimization technique on GPU
// But still acheives many fold performance gain
__global__ void calculateCFD_V1( float* input,  float* output, unsigned int Ni, unsigned int Nj, 
								   float h)
{
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; // Y - ID
	unsigned int j = threadIdx.y + blockIdx.y * blockDim.y; // X - ID

	unsigned int iPrev = i-1; // Previous Y element
	unsigned int iNext = i+1; // Next Y element

	unsigned int jPrev = j-1; //Previous X element
	unsigned int jNext = j+1; // Next X element


	unsigned int index = i * Nj + j;

	if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
		output[index] = 0.25f * (input[iPrev] + input[iNext] + input[jPrev] + input[jNext] - 4*h*h);
}
/*
//This version of Kernel uses optimization by copying the data into shared memory and hence results in better performance
__global__ void calculateCFD_V2( float* input,  float* output, unsigned int Ni, unsigned int Nj, 
								   float h)


	//Current Global ID
	int i = ; // Y - ID
	int j = ; // X - ID
	
	// Fill the size of shared memory
	__shared__ float sData [][];
	
	unsigned int index = (i)* Nj + (j) ;

	// copy data to shared memory
	sData[][] = input[]; 

	//Add synchronization. Guess Why?

	if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
		output[index] = 0.25f * (sData[][] + sData[][] + sData[][] 
			+ sData[][] - 4*h*h);

}*/