#include <cstring>
#include <stdexcept>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <vector>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.cuh"
#include "parse_graph.cuh"

#include "opt.cu"
#include "impl3.cu"
#include "impl2.cu"
#include "impl1.cu"

enum class ProcessingType {Push, Neighbor, Own, Unknown};
enum SyncMode {InCore, OutOfCore};
enum SyncMode syncMethod;
enum SmemMode {UseSmem, UseNoSmem};
enum SmemMode smemMethod;

// Open files safely.
template <typename T_file>
void openFileToAccess( T_file& input_file, std::string file_name ) {
	input_file.open( file_name.c_str() );
	if( !input_file )
		throw std::runtime_error( "Failed to open specified file: " + file_name + "\n" );
}

void outputTOFile(
	std::ofstream & outFile,
	std::vector<initial_vertex>& initGraph ){

	int glen=initGraph.size();
	int i=0;
	for(i=0;i<glen;i++){
		if (initGraph[i].vertexValue.distance==1073741824){
			outFile<< i <<":"<<"Inf."<<'\n';
			//std::cout<< i <<":"<<"Inf."<<'\n';
		}else{
			outFile<< i <<":"<<initGraph[i].vertexValue.distance<<'\n';
			//std::cout<<i <<":"<<initGraph[i].vertexValue.distance<<'\n';
		}
	}
	
	outFile.close();
}

// Execution entry point.
int main( int argc, char** argv )
{

	std::string usage =
		"\tRequired command line arguments:\n\
			Input file: E.g., --input in.txt\n\
                        Block size: E.g., --bsize 512\n\
                        Block count: E.g., --bcount 192\n\
                        Output path: E.g., --output output.txt\n\
			Processing method: E.g., --method bmf (bellman-ford), or tpe (to-process-edge), or opt (one further optimizations)\n\
			Shared memory usage: E.g., --usesmem yes, or no \n\
			Sync method: E.g., --sync incore, or outcore\n";

	try {

		std::ifstream inputFile;
		std::ofstream outputFile;
		int selectedDevice = 0;
		int bsize = 0, bcount = 0;
		//int vwsize = 32;
		//int threads = 1;
		long long arbparam = 0;
		bool nonDirectedGraph = false;		// By default, the graph is directed.
		ProcessingType processingMethod = ProcessingType::Unknown;
		syncMethod = OutOfCore;


		/********************************
		 * GETTING INPUT PARAMETERS.
		 ********************************/

		for( int iii = 1; iii < argc; ++iii )
			if ( !strcmp(argv[iii], "--method") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "bmf") )
				        processingMethod = ProcessingType::Own;
				else if ( !strcmp(argv[iii+1], "tpe") )
    				        processingMethod = ProcessingType::Neighbor;
				else if ( !strcmp(argv[iii+1], "opt") )
				    processingMethod = ProcessingType::Push;
				else{
           std::cerr << "\n Un-recognized method parameter value \n\n";
           exit(1);
         }   
			}
			else if ( !strcmp(argv[iii], "--sync") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "incore") )
				        syncMethod = InCore;
				if ( !strcmp(argv[iii+1], "outcore") )
    				        syncMethod = OutOfCore;
				else{
           std::cerr << "\n Un-recognized sync parameter value \n\n";
		   exit(1);
         }  

			}
			else if ( !strcmp(argv[iii], "--usesmem") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "yes") )
				        smemMethod = UseSmem;
				if ( !strcmp(argv[iii+1], "no") )
    				        smemMethod = UseNoSmem;
        else{
           std::cerr << "\n Un-recognized usesmem parameter value \n\n";
		   exit(1);
         }  
			}
			else if( !strcmp( argv[iii], "--input" ) && iii != argc-1 /*is not the last one*/)
				openFileToAccess< std::ifstream >( inputFile, std::string( argv[iii+1] ) );
			else if( !strcmp( argv[iii], "--output" ) && iii != argc-1 /*is not the last one*/)
				openFileToAccess< std::ofstream >( outputFile, std::string( argv[iii+1] ) );
			else if( !strcmp( argv[iii], "--bsize" ) && iii != argc-1 /*is not the last one*/)
				bsize = std::atoi( argv[iii+1] );
			else if( !strcmp( argv[iii], "--bcount" ) && iii != argc-1 /*is not the last one*/)
				bcount = std::atoi( argv[iii+1] );

		if(bsize <= 0 || bcount <= 0){
			std::cerr << "Usage: " << usage;
			exit(1);
			throw std::runtime_error("\nAn initialization error happened.\nExiting.");
		}
		if( !inputFile.is_open() || processingMethod == ProcessingType::Unknown ) {
			std::cerr << "Usage: " << usage;
			throw std::runtime_error( "\nAn initialization error happened.\nExiting." );
		}
		if( !outputFile.is_open() )
			openFileToAccess< std::ofstream >( outputFile, "out.txt" );
		CUDAErrorCheck( hipSetDevice( selectedDevice ) );
		std::cout << "Device with ID " << selectedDevice << " is selected to process the graph.\n";


		/********************************
		 * Read the input graph file.
		 ********************************/

		std::cout << "Collecting the input graph ...\n";
		std::vector<initial_vertex> parsedGraph( 0 );
		uint nEdges = parse_graph::parse(
				inputFile,		// Input file.
				parsedGraph,	// The parsed graph.
				arbparam,
				nonDirectedGraph );		// Arbitrary user-provided parameter.
		std::cout << "Input graph collected with " << parsedGraph.size() << " vertices and " << nEdges << " edges.\n";


		/********************************
		 * Process the graph.
		 ********************************/

		switch(processingMethod){
		case ProcessingType::Push:
		    pullerSortByDst(&parsedGraph, bsize, bcount);
			break;
		case ProcessingType::Own:
		    pullerSortBySrc(&parsedGraph, bsize, bcount);
		    break;
		case ProcessingType::Neighbor:
			pullerSortBySrcTPE(&parsedGraph, bsize, bcount);
			break;
		default:
			pullerSortByDst(&parsedGraph, bsize, bcount);
			break;
		}

		/********************************
		 * It's done here.
		 ********************************/
		CUDAErrorCheck( hipDeviceReset() );
		std::cout << "Done.\n";

		//ouput to file	
		outputTOFile(outputFile,parsedGraph);
		return( EXIT_SUCCESS );

	}
	catch( const std::exception& strException ) {
		std::cerr << strException.what() << "\n";
		return( EXIT_FAILURE );
	}
	catch(...) {
		std::cerr << "An exception has occurred." << std::endl;
		return( EXIT_FAILURE );
	}

}
