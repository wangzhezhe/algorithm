#include "hip/hip_runtime.h"
#include "genresult.cuh"
#include <sys/time.h>

#define MAXTHREADNUMX 1024
#define MAXTHREADNUMY 1024
//the max number of integer block number is 2147483647 (upperlimitation of integer)
#define MAXGRIDNUMX 2147483647

#define SCANNUMBER 32
#define LOADROWNUM 1024

typedef struct
{
    int row;
    int index;
} Guard;

//scan_kernal<<<blockNum, blockSize>>>(gpu_mat, gpu_vec, gpu_result, gpu_flag, gpu_base);
//shared memory -> only one pointer could access
__global__ void scan_kernal(MatrixInfo *gmat, MatrixInfo *gvec, float gresult[], int gflag[], int const gpubase)
{
    //try to modify the gresult and gflag to see the correcness of return
    int gbase = gpubase;
    //printf("test scan ... scan kernal gbase %d\n",gbase);
    int blockSize = blockDim.x;
    //int threadGlobal = gbase + blockIdx.x * blockSize + threadIdx.x;
    //int threadidKernal = blockIdx.x * blockSize + threadIdx.x;
    int threadidBlock = threadIdx.x;
    //printf("curr gbase %d, curr blockId %d, curr threadidBlock %d, curr threadidKernal %d\n",gbase,blockIdx.x,threadidBlock,threadidKernal);
    //gresult[gbase]=1.0;
    //gflag[gbase]=1;

    //init the shared memory
    //from 0-blockSize-1 store original data from the global area
    //from blockSize-2*blockSize-1 the data after scan add
    /*
    //1 size for shareMem 2*blockSize
    //2 size for mat row  1*blockSize
    //3 size for mat val  1*blockSize
    //4 size for vec val  1*blockSize
    */
    extern __shared__ float shareMem[];
    //initiallise half of the sharMem
    int blockBase = blockIdx.x * blockSize;
    float *shareAdd = (float *)shareMem;
    memcpy(shareAdd, gmat->val + gbase + blockBase, sizeof(float) * blockSize);

    int *matrow = (int *)&shareAdd[2 * blockSize];
    memcpy(matrow, gmat->rIndex + gbase + blockBase, sizeof(int) * blockSize);

    int *matcol = (int *)&matrow[blockSize];
    memcpy(matcol, gmat->cIndex + gbase + blockBase, sizeof(int) * blockSize);

    float *matval = (float *)&matcol[blockSize];
    memcpy(matval, gmat->val + gbase + blockBase, sizeof(float) * blockSize);

    //value of vector is not continuous
    float *vecval = (float *)&matval[blockSize];
    //init in parallel way
    //printf("threadidBlock %d, vector %f\n",threadidBlock,gvec->val[matcol[threadidBlock]]);
    vecval[threadidBlock] = gvec->val[matcol[threadidBlock]];



    //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
    //{
    //printf("gbase %d blockid %d tidblock %d, tidkernal %d , tidglobal %d,  sharadd %f, matrow %d, matcol %d matval %f, vecval %f\n",
    //         gbase, blockIdx.x, threadidBlock, threadidKernal, threadGlobal, shareAdd[threadidBlock], matrow[threadidBlock], matcol[threadidBlock], matval[threadidBlock], vecval[threadidBlock]);
    //}
    int span = 1;
    float product = 0;
    float addValue = 0;
    //compute
    product = 1.0 * shareAdd[threadidBlock] * vecval[threadidBlock];
    //shareAdd[threadidBlock] = product;
    atomicExch(&shareAdd[threadidBlock],product);
    __syncthreads();
    
    //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
    //{
    //    printf("debug step1 id %d threadidBlock %f \n", threadGlobal, shareAdd[threadidBlock]);
    //}

    for (span = 1; span < blockSize; span *= 2)
    {
        //add operation

        if (threadidBlock >= span && matrow[threadidBlock] == matrow[threadidBlock - span])
        {
            addValue = shareAdd[threadidBlock] + shareAdd[threadidBlock - span];
            atomicExch(&shareAdd[blockSize + threadidBlock], addValue);

            //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
            //{
            //    printf("debug step2 id %d threadidBlock %f \n", threadGlobal, shareAdd[threadidBlock]);
            //}
        }
        else
        {
            //same with original one, do not add
            atomicExch(&shareAdd[blockSize + threadidBlock], shareAdd[threadidBlock]);
        }

        //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
        //{
        //    printf("debug step3 id %d threadidBlock %f \n", threadGlobal, shareAdd[threadidBlock]);
        //}

        __syncthreads();

        shareAdd[threadidBlock] = shareAdd[blockSize + threadidBlock];

        //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
        //{
        //    printf("debug step4 id %d threadidBlock %f \n", threadGlobal, shareAdd[threadidBlock]);
        //}

        __syncthreads();
    }

    //computed the final result, label at the end position of the block
    //check curr shareADD
    //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
    //{
    //    printf("debugbeforegresult %f globalid %d\n", shareAdd[threadidBlock], threadGlobal);
    //}
    atomicExch(&gresult[gbase + blockBase + threadidBlock], shareAdd[threadidBlock]);
    atomicExch(&gflag[gbase + blockBase + blockSize - 1], 1);
    //if (threadGlobal == debugtid1 || threadGlobal == debugtid2 || matrow[threadidBlock]==debugrnum)
    //{
    //    printf("debug globalid %d gbase %d gflag label %d gflag value %d\n", threadGlobal, gbase, gbase + blockBase + blockSize - 1, gflag[gbase + blockBase + blockSize - 1]);
    //}
}

void matSwap(MatrixInfo *mat, int i, int j)
{
    int tempr;
    int tempc;
    float tempv;

    tempr = mat->rIndex[j];
    mat->rIndex[j] = mat->rIndex[i];
    mat->rIndex[i] = tempr;
    //change cIndex
    tempc = mat->cIndex[j];
    mat->cIndex[j] = mat->cIndex[i];
    mat->cIndex[i] = tempc;
    //change val
    tempv = mat->val[j];
    mat->val[j] = mat->val[i];
    mat->val[i] = tempv;
}

int partition(MatrixInfo *mat, int low, int high)
{
    int pivot = mat->rIndex[high]; // pivot
    int i = (low - 1);             // Index of smaller element

    for (int j = low; j <= high - 1; j++)
    {
        // If current element is smaller than or
        // equal to pivot
        if (mat->rIndex[j] <= pivot)
        {
            i++; // increment index of smaller element
            matSwap(mat, i, j);
        }
    }
    matSwap(mat, i + 1, high);
    return (i + 1);
}

void matrixSortQuick(MatrixInfo *mat, int low, int high)
{

    if (low < high)
    {
        /* pi is partitioning index, arr[p] is now
           at right place */
        int pi = partition(mat, low, high);
        printf("qsort %d\n", pi);
        // Separately sort elements before
        // partition and after partition
        matrixSortQuick(mat, low, pi - 1);
        matrixSortQuick(mat, pi + 1, high);
    }
}
void matrixSortQuick2(MatrixInfo *mat, int left, int right)
{
    int i = left, j = right;
    int pivot = mat->rIndex[(left + right) / 2];

    /* partition */
    while (i <= j)
    {
        while (mat->rIndex[i] < pivot)
            i++;
        while (mat->rIndex[j] > pivot)
            j--;
        if (i <= j)
        {
            matSwap(mat, i, j);
            i++;
            j--;
        }
    };

    /* recursion */
    if (left < j)
        matrixSortQuick2(mat, left, j);
    if (i < right)
        matrixSortQuick2(mat, i, right);
}

void matrixSort(MatrixInfo *mat)
{
    int elemNum = mat->nz;

    //sort according the mat->M
    int i, j;
    int tempr;
    int tempc;
    float tempv;
    for (i = 0; i < elemNum; i++)
    {
        for (j = i + 1; j < elemNum; j++)
        {
            if (mat->rIndex[j] < mat->rIndex[i])
            {
                tempr = mat->rIndex[j];
                mat->rIndex[j] = mat->rIndex[i];
                mat->rIndex[i] = tempr;
                //change cIndex
                tempc = mat->cIndex[j];
                mat->cIndex[j] = mat->cIndex[i];
                mat->cIndex[i] = tempc;
                //change val
                tempv = mat->val[j];
                mat->val[j] = mat->val[i];
                mat->val[i] = tempv;
            }
        }
    }
    return;
}

int getGuardRecord(MatrixInfo *mat, Guard **gRecord)
{
    int elem_num = mat->nz;
    int i;
    int gindex = 0;
    for (i = 1; i < elem_num; i++)
    {
        if (mat->rIndex[i] != mat->rIndex[i - 1])
        {

            (*gRecord)[gindex].index = i - 1;
            (*gRecord)[gindex].row = mat->rIndex[i - 1];
            //printf("ri %d ri-1 %d guard %d i-1 %d gindex %d\n",mat->rIndex[i],mat->rIndex[i-1],(*gRecord)[gindex].index,i-1,gindex);
            gindex++;
        }
    }
    //last one
    (*gRecord)[gindex].index = elem_num - 1;
    (*gRecord)[gindex].row = mat->rIndex[elem_num - 1];
    return (gindex + 1);
}

//glen is the length of the guard
//return cloest guard position
int getCloseGuardIndex(int currIndex, Guard *guard, int glen)
{

    int i = 0;
    int r = -2;
    for (i = 0; i < glen - 1; i++)
    {
        if (currIndex == guard[i].index || currIndex == guard[i + 1].index)
        {
            return -1;
        }
        else
        {
            if (currIndex > guard[i].index && currIndex < guard[i + 1].index)
            {
                return guard[i + 1].index;
            }
        }
    }

    printf("debugclose currIndex %d\n", currIndex);
    return r;
}

//make sure if the index equal to the element in guard
//range the element in guard to make sure if the index is in the array
int checkGuard(int index, int *guard, int len)
{

    int i = 0;
    int *temp = guard;

    for (i = 0; i < len; i++)
    {
        if (index == *temp)
        {
            return 1;
        }
    }
    return 0;
}

/*input parameters:
mat: sparse matrix M in coo format
vec: input vector V in coo frmat
res: the multiplication of M*V

*/
int getMulScan(MatrixInfo *mat, MatrixInfo *vec, MatrixInfo *finalres, int blockSize, int blockNum)
{

    struct timespec prestart, preend;

    clock_gettime(CLOCK_MONOTONIC_RAW, &prestart);


    //check parameters
    printf("mulScan matrix m n nz %d %d %d\n", mat->M, mat->N, mat->nz);
    printf("mulScan vector m n nz %d %d %d\n", vec->M, vec->N, vec->nz);

    int errCode = 1;
    if (mat == NULL || vec == NULL || finalres == NULL)
    {
        errCode = 2;
        printf("getMulScan err, mat , vec, res could not be null\n");
        return errCode;
    }

    if (blockSize > (MAXTHREADNUMX * MAXTHREADNUMX))
    {
        errCode = 2;
        printf("parameters err, blockSize (%d) is larger than limitation\n", blockSize);
        return errCode;
    }

    //if (mat->nz > LOADROWNUM)
    //{
    //TODO divide logic into different parts for large size matrix
    //    errCode = 2;
    //    printf("matrix size (%d) is larger than upper limitation\n", mat->nz);
    //    return errCode;
    // }

    printf("non-zero number %d\n", mat->nz);

    //original mat value
    int i;
    for (i = 0; i < mat->nz; i++)
    {
        //printf("indexoriginal %d r %d c %d val %f\n",i, mat->rIndex[i], mat->cIndex[i], mat->val[i]);
        if(mat->rIndex[i]<0 || mat->cIndex[i]<0){
            printf("matrix load fail: \nindexoriginal %d r %d c %d val %f\n",i, mat->rIndex[i], mat->cIndex[i], mat->val[i]);
            exit(1);
        }
    }

    /*
    sorting by column
    */

    //matrixSort(mat);
    //matrixSortQuick(mat,0,mat->nz-1);
    matrixSortQuick2(mat, 0, mat->nz - 1);

    /*
    summary the matrix, get guardRecord, length= mat->M*sizeof(int)
    */

    Guard *guard_record = (Guard *)malloc((mat->M + 1) * sizeof(Guard));

    int glength = getGuardRecord(mat, &guard_record);

    //check guard

    //for (i = 0; i < glength; i++)
    //{
    //    printf("index %d guardPosition %d guard rowid %d\n", i, guard_record[i].index,guard_record[i].row);
    //}

    //check sort

    //for (i = 0; i < mat->nz; i++)
    //{
    //    printf("index %d r %d c %d val %f\n", i, mat->rIndex[i], mat->cIndex[i], mat->val[i]);
    //}

    printf("Sorting by row number finished\n");

    /*Allocate things...*/

    //malloc innter pointer value
    //refer https://github.com/parallel-forall/code-samples/blob/master/posts/unified-memory/dataElem_um.cu
    // void** is really important here to prevent the error like all cuda device is busy

    MatrixInfo *gpu_mat;
    MatrixInfo *gpu_vec;
    float *gpu_result;
    int *gpu_flag;
    int *gpu_base;

    hipMallocManaged((void **)&gpu_mat, sizeof(MatrixInfo));
    hipMallocManaged((void **)&gpu_vec, sizeof(MatrixInfo));

    hipMallocManaged((void **)&(gpu_mat->rIndex), sizeof(int) * (mat->nz));
    memcpy(gpu_mat->rIndex, mat->rIndex, sizeof(int) * mat->nz);

    hipMallocManaged((void **)&(gpu_mat->cIndex), sizeof(int) * (mat->nz));
    memcpy(gpu_mat->cIndex, mat->cIndex, sizeof(int) * mat->nz);

    hipMallocManaged((void **)&(gpu_mat->val), sizeof(float) * (mat->nz));
    memcpy(gpu_mat->val, mat->val, sizeof(float) * mat->nz);

    gpu_mat->M = mat->M;
    gpu_mat->N = mat->N;
    gpu_mat->nz = mat->nz;

    //vector assignment
    gpu_vec->rIndex = NULL;
    gpu_vec->cIndex = NULL;

    //only val is useful here, other value is null for vector
    hipMallocManaged((void **)&(gpu_vec->val), sizeof(float) * (vec->nz));
    memcpy(gpu_vec->val, vec->val, sizeof(float) * vec->nz);

    //result

    hipMallocManaged((void **)&gpu_result, sizeof(float) * (mat->nz));
    hipMallocManaged((void **)&gpu_flag, sizeof(int) * (mat->nz));

    int *flag = (int *)calloc(mat->nz, sizeof(int));
    float *result = (float *)calloc(mat->nz, sizeof(float));

    //still need to initialise the gpu memory even if the memory is just allocated
    memcpy(gpu_flag, flag, mat->nz * sizeof(int));
    memcpy(gpu_result, result, mat->nz * sizeof(float));

    //base
    hipMallocManaged((void **)&(gpu_base), sizeof(int));

    /* Start Execution */

    printf("allocation things on gpu ok\n");
    printf("block size (%d),block number (%d)\n", blockSize, blockNum);


    clock_gettime(CLOCK_MONOTONIC_RAW, &preend);


    struct timespec start, end;

    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    int all_elem_num = mat->nz;
    int workload = blockNum * blockSize;
    int iteration = all_elem_num / workload;
    int reminder_load = all_elem_num % workload;

    printf("workload %d iteration %d reminder %d\n", workload, iteration, reminder_load);

    //int *base = (int *)malloc(sizeof(int));
    //base[0] = 0;
    int base = 0;
    //trick gsharemem space array

    //1 size for shareMem 2*blockSize
    //2 size for mat row  1*blockSize
    //3 size for mat val  1*blockSize
    //4 size for vec val  1*mat->M

    int sharSize = blockSize * (3 * sizeof(float) + 2 * sizeof(int)) + blockSize * (sizeof(float));
    //int rowNum=mat->M;
    i = 0;
    for (i = 0; i < iteration; i++)
    {
        base = workload * i;
        //hipMemcpy(gpu_base, base, sizeof(int), hipMemcpyHostToDevice);
        //printf("input base iteration %d\n",base);
        scan_kernal<<<blockNum, blockSize, sharSize>>>(gpu_mat, gpu_vec, gpu_result, gpu_flag, base);
        //hipDeviceSynchronize();
    }
    if (reminder_load > 0)
    {
        int newblocknum = reminder_load / blockSize;
        int newreminder = reminder_load % blockSize;
        printf("newblocknum %d newreminder %d\n", newblocknum, newreminder);

        base = workload * i;

        if (newblocknum > 0)
        {
            //printf("input base iteration %d\n",base);
            scan_kernal<<<newblocknum, blockSize, sharSize>>>(gpu_mat, gpu_vec, gpu_result, gpu_flag, base);
        }
        if (newreminder > 0)
        {
            //less than one block
            int newbase = base + newblocknum * blockSize;
            int reminderSize = newreminder * (3 * sizeof(float) + 2 * sizeof(int)) + newreminder * (sizeof(float));
            printf("input reminder iteration %d\n", newbase);
            scan_kernal<<<1, newreminder, reminderSize>>>(gpu_mat, gpu_vec, gpu_result, gpu_flag, newbase);
        }

        //gpu_base = base;
        //hipMemcpy(gpu_base, base, sizeof(int), hipMemcpyHostToDevice);
    }

    //sum up from guard, flag, result
    //copy result and flag from gpu
    hipMemcpy(flag, gpu_flag, mat->nz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result, gpu_result, mat->nz * sizeof(float), hipMemcpyDeviceToHost);

    //check flag, result, ok here, the result could be write back into the cpu
    //for (i = 0; i < mat->nz; i++)
    //{
    //    printf("index %d flag %d  raw result %f\n", i, flag[i], result[i]);
    //}

    //TODO improve the parallelism ability here

    int closeIndex = 0;
    for (i = 0; i < all_elem_num; i++)
    {
        if (flag[i] == 1)
        {

            //closeIndex = getCloseGuardIndex(i, guard_record, mat->M);
            closeIndex = getCloseGuardIndex(i, guard_record, glength);
            //printf("debug curr index %d closeIndex %d\n",i,closeIndex);
            if (closeIndex >= 0)
            {
                result[closeIndex] = result[closeIndex] + result[i];
            }
            if (closeIndex == -2)
            {
                printf("bug in get cloestguard\n");
                exit(1);
            }
        }
    }

    Guard *temp = guard_record;
    int nzindex = 0;
    int finalindex = 0;
    int padnum = 0;
    for (i = 0; i < all_elem_num; i++)
    {
        if (i == temp[nzindex].index)
        {
            if (nzindex == 0)
            {
                if (temp[nzindex].row > 0)
                {
                    padnum = temp[nzindex].row;
                    while (padnum > 0)
                    {
                        finalres->val[finalindex] = 0;
                        finalindex++;
                        padnum--;
                    }
                }
            }
            else if (nzindex >= 1)
            {
                padnum = temp[nzindex].row - temp[nzindex - 1].row;
                while ((padnum - 1) > 0)
                {
                    finalres->val[finalindex] = 0;
                    finalindex++;
                    padnum--;
                }
            }
            finalres->val[finalindex] = result[i];
            finalindex++;

            //add zero here
            nzindex++;
        }
    }

    //print the final result res
    //for (i = 0; i < mat->M; i++)
    //{
    //    printf("final lnum %d result %f\n", i, finalres->val[i]);
    //}

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    
    printf("Preprocess Time: %lu milli-seconds\n", 1000 * (preend.tv_sec - prestart.tv_sec) + (preend.tv_nsec - prestart.tv_nsec) / 1000000);
    printf("Kernel Time: %lu milli-seconds\n", 1000 * (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1000000);

    /*Deallocate, please*/
    //hipDeviceReset();
    free(guard_record);
    free(result);
    free(flag);

    //be careful for bus error if do the following operation first
    //hipFree(gpu_mat->rIndex);
    //hipFree(gpu_mat->cIndex);
    //hipFree(gpu_mat->val);

    hipFree(gpu_mat);
    hipFree(gpu_vec);

    //return 1 if everything is ok
    //printf("curr errCode %d\n", errCode);
    return errCode;
}
